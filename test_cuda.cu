#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello_cuda() {
    printf("Hello from GPU thread %d!\n", threadIdx.x);
}

int main() {
    printf("CUDA Test Program\n");
    
    // GPU 정보 확인
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);
    
    if (deviceCount > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("Device 0: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Global memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        
        // 간단한 커널 실행
        hello_cuda<<<1, 5>>>();
        hipDeviceSynchronize();
    }
    
    printf("CUDA test completed!\n");
    return 0;
} 