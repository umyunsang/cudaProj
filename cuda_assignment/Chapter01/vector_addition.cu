
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

//basically just fills the array with index.
void fill_array(int *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}
int main(void) {
	int *a, *b, *c;
	int size = N * sizeof(int);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a);
	b = (int *)malloc(size); fill_array(b);
	c = (int *)malloc(size);

	host_add(a,b,c);

	print_output(a,b,c);

	free(a); free(b); free(c);


	return 0;
} 