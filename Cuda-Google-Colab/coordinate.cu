
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printCoordinate(int *A,const int nx,const int ny)
{
  int ix=threadIdx.x+blockIdx.x*blockDim.x;
  int iy=threadIdx.y+blockIdx.y*blockDim.y;
  unsigned int idx=iy*nx+ix;
  printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d)"
          "global index %2d ival %2d\n",threadIdx.x,threadIdx.y,
          blockIdx.x,blockIdx.y,ix,iy,idx,A[idx]);
}

int main(int argc,char ** argv)
{
  hipSetDevice(0);
  int nx = 8;
  int ny = 6;
  // Malloc
  int* A_host = (int*)malloc(nx*ny*sizeof(int));
  for(int i = 0; i<nx*ny; ++i){
      A_host[i] = 0;
  }

  // cudaMalloc
  int *A_dev=NULL;
  hipMalloc((void**)&A_dev, nx*ny*sizeof(int));
  hipMemcpy(A_dev, A_host, nx*ny*sizeof(int), hipMemcpyHostToDevice);

  dim3 block(4,2);
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);

  printCoordinate<<<grid,block>>>(A_dev,nx,ny);
  hipDeviceSynchronize();

  // Free
  hipFree(A_dev);
  free(A_host);

  hipDeviceReset();
  return 0;
}
