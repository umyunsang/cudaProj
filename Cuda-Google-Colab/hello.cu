
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello(void)
{
  printf("GPU: Hello!\n");
}
int main(int argc,char **argv)
{
  printf("CPU: Hello!\n");
  hello<<<1,10>>>();
  hipDeviceReset();
  return 0;
}
